#include <stdio.h>
#include <hip/hip_runtime.h>

const int N = 256;  // size of the square matrix

__global__ void matrixMul(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    for (int i = 0; i < N; i++) {
        sum += a[row * N + i] * b[i * N + col];
    }
    c[row * N + col] = sum;
}

int main() {
    int *h_a, *h_b, *h_c;  // host matrices
    int *d_a, *d_b, *d_c;  // device matrices

    size_t size = N * N * sizeof(int);

    // Allocate space on the host
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_a[i] = rand() % 1024;
        h_b[i] = rand() % 1024;
    }

    // Allocate space on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    dim3 numBlocks(1, 1);
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy result matrix back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print a few results
    printf("Sample results:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

